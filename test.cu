#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <hip/hip_runtime.h>

#define TGIG 536870912
#define SLEEP_TIME 100

int main() {
    float N = TGIG*3.2; 
    float *host_array = (float*)malloc(N * sizeof(float));
    if (!host_array) {
        printf("Host malloc failed\n");
        return 1;
    }

    for (size_t i = 0; i < N; ++i) host_array[i] = (float)i;

    float *device_array;
    hipError_t err = hipMalloc((void**)&device_array, N * sizeof(float));
    if (err != hipSuccess) {
        printf("hipMalloc failed: %s\n", hipGetErrorString(err));
        free(host_array);
        return 1;
    }

    err = hipMemcpy(device_array, host_array, N * sizeof(float), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        printf("hipMemcpy failed: %s\n", hipGetErrorString(err));
        hipFree(device_array);
        free(host_array);
        return 1;
    }

    free(host_array);
    printf("Allocated and copied %f GB to GPU. Sleeping for %d seconds...\nWill Now start deallocating from RAM\n", (N*4)/1000000000, SLEEP_TIME);
    sleep(SLEEP_TIME);
    hipFree(device_array);
    return 0;
}
